#include <stdio.h>

// Helper function to convert SM version to number of cores
int _ConvertSMVer2Cores(int major, int minor)
{
    // Defpth of each generation is only the major version # and each arch is the same or better than its predecessors
    // Only items which have been officially announced are included in this list
    typedef struct {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    { { 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
      { 0x32, 192 }, // Kepler Generation (SM 3.2) GK11x class
      { 0x35, 192 }, // Kepler Generation (SM 3.5) GK21x class
      { 0x37, 192 }, // Kepler Generation (SM 3.7) GK21x class
      { 0x50, 128 }, // Maxwell Generation (SM 5.0) GM10x class
      { 0x52, 128 }, // Maxwell Generation (SM 5.2) GM20x class
      { 0x53, 128 }, // Maxwell Generation (SM 5.3) GM20x class
      { 0x60,  64 }, // Pascal Generation (SM 6.0) GP100 class
      { 0x61, 128 }, // Pascal Generation (SM 6.1) GP10x class
      { 0x62, 128 }, // Pascal Generation (SM 6.2) GP10x class
      { 0x70,  64 }, // Volta Generation (SM 7.0) GV100 class
      { 0x72,  64 }, // Turing Generation (SM 7.2) TU102 class
      { 0x75,  64 }, // Turing Generation (SM 7.5) TU116 class
      { 0x80,  64 }, // Ampere Generation (SM 8.0) GA100 class
      { 0x86, 128 }, // Ampere Generation (SM 8.6) GA104 class
      { 0x89, 128 }, // Ada Lovelace Generation (SM 8.9) AD102 class
      {   -1, -1 }
    };

    int index = 0;
    while (nGpuArchCoresPerSM[index].SM != -1) {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }
    // If we don't find the values, we default use the previous one to run properly
    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
    return nGpuArchCoresPerSM[index-1].Cores;
}

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Maximum number of threads per SM: %d\n", devProp.maxThreadsPerMultiProcessor);

    // Add detailed core information
    printf("\nCore Information per SM:\n");
    printf("FP32 (CUDA) cores:            %d\n", _ConvertSMVer2Cores(devProp.major, devProp.minor));
    printf("FP16 cores:                   %d\n", _ConvertSMVer2Cores(devProp.major, devProp.minor) * 2);  // FP16 is typically 2x FP32
    printf("INT32 cores:                  %d\n", _ConvertSMVer2Cores(devProp.major, devProp.minor));      // Usually same as FP32
    
    // Tensor cores vary by architecture
    int tensorCoresPerSM = 0;
    if (devProp.major >= 7) {  // Volta and newer
        if (devProp.major == 7 && devProp.minor == 0) {  // Volta
            tensorCoresPerSM = 8;
        } else if (devProp.major == 7 && devProp.minor >= 2) {  // Turing
            tensorCoresPerSM = 8;
        } else if (devProp.major == 8) {  // Ampere
            tensorCoresPerSM = 4;
        } else if (devProp.major == 9) {  // Hopper
            tensorCoresPerSM = 4;
        }
    }
    printf("Tensor cores:                 %d\n", tensorCoresPerSM);
    
    printf("\nTotal Cores across all SMs:\n");
    printf("Total FP32 cores:             %d\n", _ConvertSMVer2Cores(devProp.major, devProp.minor) * devProp.multiProcessorCount);
    printf("Total FP16 cores:             %d\n", _ConvertSMVer2Cores(devProp.major, devProp.minor) * 2 * devProp.multiProcessorCount);
    printf("Total INT32 cores:            %d\n", _ConvertSMVer2Cores(devProp.major, devProp.minor) * devProp.multiProcessorCount);
    printf("Total Tensor cores:           %d\n", tensorCoresPerSM * devProp.multiProcessorCount);
    
    printf("\nKernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ?"Yes" : "No"));
    return;
}
 
int main()
{
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);
 
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }
    return 0;
}
